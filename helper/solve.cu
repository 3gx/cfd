
#include <hip/hip_runtime.h>
// Doolittle uses unit diagonals for the lower triangle
void Doolittle(int d,double*S,double*D){
   for(int k=0;k<d;++k){
      for(int j=k;j<d;++j){
         double sum=0.;
         for(int p=0;p<k;++p)sum+=D[k*d+p]*D[p*d+j];
         D[k*d+j]=(S[k*d+j]-sum); // not dividing by diagonals
      }
      for(int i=k+1;i<d;++i){
         double sum=0.;
         for(int p=0;p<k;++p)sum+=D[i*d+p]*D[p*d+k];
         D[i*d+k]=(S[i*d+k]-sum)/D[k*d+k];
      }
   }
}
void solveDoolittle(int d,double*LU,double*b,double*x){
   double y[d];
   for(int i=0;i<d;++i){
      double sum=0.;
      for(int k=0;k<i;++k)sum+=LU[i*d+k]*y[k];
      y[i]=(b[i]-sum); // not dividing by diagonals
   }
   for(int i=d-1;i>=0;--i){
      double sum=0.;
      for(int k=i+1;k<d;++k)sum+=LU[i*d+k]*x[k];
      x[i]=(y[i]-sum)/LU[i*d+i];
   }
}
